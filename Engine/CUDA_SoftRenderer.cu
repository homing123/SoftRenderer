#include "hip/hip_runtime.h"
#include "EngineLib.h"
#include "CUDA_Define.cuh"

const int h_MinThreadCount = 32;
const int h_MaxThreadCount = 512; // 1024 ��� �ϴ��� 1024�ϴϱ� ����ȵƾ���
const int h_MaxBlockCountX = 1024;
const int h_MaxBlockCountY = 1024;
const int h_ThreadMulBlockX = h_MaxThreadCount * h_MaxBlockCountX;
const int h_MaxTotalCount = h_MaxThreadCount * h_MaxBlockCountX * h_MaxBlockCountY;
float* h_Log = new float[100];
float* d_Log;
#pragma region �����Լ�
const int MatSize = sizeof(float4) * 4;

//���ͳݿ��� �ܾ�°� �ȿ� ���ư��� ���� �˾ƺ� �ʿ� ����
__device__ static float atomicMax(float* address, float val)
{
	int* address_as_i = (int*)address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
			__float_as_int(::fmaxf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}

__device__ static float atomicMin(float* address, float val)
{
	int* address_as_i = (int*)address;
	int old = *address_as_i, assumed;
	do {
		assumed = old;
		old = ::atomicCAS(address_as_i, assumed,
			__float_as_int(::fminf(val, __int_as_float(assumed))));
	} while (assumed != old);
	return __int_as_float(old);
}


//���°��� �ִ��� ���Ϲ�� ã�ƾ���
void Block_Thread_Size(const int total_Size, dim3& out_Block, int& out_Thread)
{
	if (total_Size > h_MaxTotalCount)
	{
		int a = 0;
		return;
	}
	else if (total_Size >= h_ThreadMulBlockX)
	{
		out_Block.z = 1;
		if (total_Size == h_ThreadMulBlockX)
		{
			out_Block.y = total_Size / h_ThreadMulBlockX;
		}
		else
		{
			out_Block.y = total_Size / h_ThreadMulBlockX + 1;
		}
		out_Block.x = h_MaxBlockCountX;
		out_Thread = h_MaxThreadCount;
	}
	else if (total_Size >= h_MaxThreadCount)
	{
		out_Block.z = 1;
		out_Block.y = 1;
		if (total_Size == h_MaxThreadCount)
		{
			out_Block.x = total_Size / h_MaxThreadCount;
		}
		else
		{
			out_Block.x = total_Size / h_MaxThreadCount + 1;
		}
		out_Thread = h_MaxThreadCount;
	}
	else
	{
		out_Block.z = 1;
		out_Block.y = 1;
		out_Block.x = 1;
		out_Thread = (((total_Size >> 5) + 1) << 5);
	}
}

//Ÿ�ϸ� ��� �Ẹ��
__constant__ int MatMulMatCount;
__constant__ float4 ConstantMat[4];

__global__ void ConstantMatMulMat(d_Matrix4x4* s_m, d_Matrix4x4* d_m)
{
	//extern __shared__ float rightMat[];
	int idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx >= MatMulMatCount)
	{
		return;
	}
	int matIdx = idx >> 4;
	int eleIdx = idx & 15;
	int _y = eleIdx >> 2; //warp = 32 thread, �� ������ _y���� �����ϴ�
	int _x = eleIdx & 3;
	float4 leftf4 = ConstantMat[_y];

	//tiling ������
	d_m[matIdx].Ele[eleIdx] = leftf4.x * s_m[matIdx].Ele[_x] + leftf4.y * s_m[matIdx].Ele[_x + 4] + leftf4.z * s_m[matIdx].Ele[_x + 8] + leftf4.w * s_m[matIdx].Ele[_x + 12];

	//tiling ����
	/*rightMat[threadIdx.x] = s_m[matIdx].Ele[eleIdx];
	__syncthreads();
	int mulValue = (matIdx * 16) & (blockDim.x - 1);
	d_m[matIdx].Ele[eleIdx] = leftf4.x * rightMat[mulValue + _x] + leftf4.y * rightMat[mulValue + _x + 4] + leftf4.z * rightMat[mulValue + _x + 8] + leftf4.w * rightMat[mulValue + _x + 12];*/

}
hipStream_t S_ConstantMatMulMat;
void CU_ConstantMatMulMat(int count, Matrix4x4& h_mat, d_Matrix4x4* s_m, d_Matrix4x4* d_m)
{
	hipMemcpyToSymbolAsync(HIP_SYMBOL(ConstantMat), &h_mat, MatSize, 0, hipMemcpyHostToDevice, S_ConstantMatMulMat);
	int h_matmulCount = count * 16;
	hipMemcpyToSymbolAsync(HIP_SYMBOL(MatMulMatCount), &h_matmulCount, sizeof(int), 0, hipMemcpyHostToDevice, S_ConstantMatMulMat);

	dim3 block;
	int thread;
	Block_Thread_Size(count * 16, block, thread);
	//tiling ������
	ConstantMatMulMat << <block, thread, thread * sizeof(float), S_ConstantMatMulMat >> > (s_m, d_m);

	//tiling ����
	//ConstantMatMulMat << <block, thread,0, S_ConstantMatMulMat >> > (s_m, d_m);

	hipStreamSynchronize(S_ConstantMatMulMat);
}
__constant__ int KoggeStoneCount_Block;

//0, 511, 1023, 1535 ...~ ������ ������ ���ش�.
__global__ void KoggeStoneScan_blocks(int* d_res, int* d_blockAddValue)
{
	extern __shared__ int shared[];
	int idx = threadIdx.x;
	if (idx >= KoggeStoneCount_Block)
	{
		return;
	}
	if (idx == 0)
	{
		shared[idx] = 0;
	}
	else
	{
		shared[idx] = d_res[511 + 512 * (idx - 1)];
	}
	for (int stride = 1; stride < blockDim.x; stride <<= 1)
	{
		__syncthreads();
		if (threadIdx.x >= stride)
		{
			shared[threadIdx.x] += shared[threadIdx.x - stride];
		}
	}

	d_blockAddValue[idx] = shared[threadIdx.x];
}

//������ �� ���� �����ش�.
__global__ void KoggeStoneScan_int_Final(int* d_res, int* d_blockAddValue, int count)
{
	int idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx >= count)
	{
		return;
	}
	int bIdx = gridDim.x * blockIdx.y + blockIdx.x;
	d_res[idx] += d_blockAddValue[bIdx];
}


void CU_KoggeStoneScan_Block(int* d_res, dim3 block, int thread, int count, int blockCount, hipStream_t& stream)
{
	hipMemcpyToSymbolAsync(HIP_SYMBOL(KoggeStoneCount_Block), &blockCount, sizeof(int), 0, hipMemcpyHostToDevice, stream);
	int blockthread = (((blockCount >> 5) + 1) << 5);

	int* d_AddValue;
	hipMalloc(&d_AddValue, sizeof(int) * blockCount);
	KoggeStoneScan_blocks << <1, blockthread, blockthread * sizeof(int), stream >> > (d_res, d_AddValue);
	KoggeStoneScan_int_Final << <block, thread, 0, stream >> > (d_res, d_AddValue, count);
	hipFree(d_AddValue);
}
#pragma endregion


int* d_TexBufferStartIdx;
int* d_TexBufferCount;

int* d_TexWidth;
int* d_TexHeight;

d_Color32* d_TexBuffer;

int* d_MeshVertexStartIdx;
int* d_MeshVertexCount;

int* d_MeshIndexStartIdx;
int* d_MeshIndexCount;

float3* d_MeshVertices;
float2* d_MeshUVs;
int* d_MeshIndices;

float3* d_BvBox_Min;
float3* d_BvBox_Max;

int h_CurUploadTextureCount = 0;
int h_CurUploadTexBufferCount = 0;

int h_CurUploadMeshCount = 0;
int h_CurUploadVertexCount = 0;
int h_CurUploadIndexCount = 0;
std::map<int, int> h_MeshTriangleCountLookUpTable;
hipStream_t S_BGDepthInit;

void CU_Init(int totalMeshCount, int totalTextureCount, int totalVertexCount, int totalIndexCount, int totalTexBufferCount)
{
	hipStreamCreate(&S_BGDepthInit);
	hipStreamCreate(&S_ConstantMatMulMat);
	hipMalloc(&d_Log, sizeof(float) * 100);

	hipMalloc(&d_TexBufferStartIdx, sizeof(int) * totalTextureCount);
	hipMalloc(&d_TexBufferCount, sizeof(int) * totalTextureCount);
	hipMalloc(&d_TexWidth, sizeof(int) * totalTextureCount);
	hipMalloc(&d_TexHeight, sizeof(int) * totalTextureCount);

	hipMalloc(&d_TexBuffer, sizeof(d_Color32) * totalTexBufferCount);

	hipMalloc(&d_MeshVertexStartIdx, sizeof(int) * totalMeshCount);
	hipMalloc(&d_MeshVertexCount, sizeof(int) * totalMeshCount);
	hipMalloc(&d_MeshIndexStartIdx, sizeof(int) * totalMeshCount);
	hipMalloc(&d_MeshIndexCount, sizeof(int) * totalMeshCount);

	hipMalloc(&d_MeshVertices, sizeof(float3) * totalVertexCount);
	hipMalloc(&d_MeshUVs, sizeof(float2) * totalVertexCount);
	hipMalloc(&d_MeshIndices, sizeof(int) * totalIndexCount);
	hipMalloc(&d_BvBox_Min, sizeof(float3) * totalMeshCount);
	hipMalloc(&d_BvBox_Max, sizeof(float3) * totalMeshCount);
}

int CU_AddTexture(Texture* pTexture)
{
	int bufferSize = pTexture->GetSize();
	int width = pTexture->GetWidth();
	int height = pTexture->GetHeight();
	hipMemcpy(d_TexBufferStartIdx + h_CurUploadTextureCount, &h_CurUploadTexBufferCount, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_TexBufferCount + h_CurUploadTextureCount, &bufferSize, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_TexWidth + h_CurUploadTextureCount, &width, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_TexHeight + h_CurUploadTextureCount, &width, sizeof(int), hipMemcpyHostToDevice);

	hipMemcpy(d_TexBuffer + h_CurUploadTexBufferCount, pTexture->GetBuffer(), sizeof(d_Color32) * bufferSize, hipMemcpyHostToDevice);

	h_CurUploadTextureCount++;
	h_CurUploadTexBufferCount += bufferSize;

	return h_CurUploadTextureCount - 1;
}

int CU_AddMesh(Mesh* pMesh)
{
	int vertexCount = pMesh->VertexCount();
	int indexCount = pMesh->IndexCount();

	h_MeshTriangleCountLookUpTable[h_CurUploadMeshCount] = indexCount / 3;

	hipMemcpy(d_MeshVertexStartIdx + h_CurUploadMeshCount, &h_CurUploadVertexCount, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_MeshVertexCount + h_CurUploadMeshCount, &vertexCount, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_MeshVertices + h_CurUploadVertexCount, pMesh->GetVertices(), sizeof(float3) * vertexCount, hipMemcpyHostToDevice);
	hipMemcpy(d_MeshUVs + h_CurUploadVertexCount, pMesh->GetUVs(), sizeof(float2) * vertexCount, hipMemcpyHostToDevice);

	hipMemcpy(d_MeshIndexStartIdx + h_CurUploadMeshCount, &h_CurUploadIndexCount, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_MeshIndexCount + h_CurUploadMeshCount, &indexCount, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_MeshIndices + h_CurUploadIndexCount, pMesh->GetIndices(), sizeof(int) * indexCount, hipMemcpyHostToDevice);

	Vector3 bv_min, bv_max;
	pMesh->GetpBV()->GetBoxInfo(bv_min, bv_max);
	hipMemcpy(d_BvBox_Max + h_CurUploadMeshCount, &bv_max, sizeof(float3), hipMemcpyHostToDevice);
	hipMemcpy(d_BvBox_Min + h_CurUploadMeshCount, &bv_min, sizeof(float3), hipMemcpyHostToDevice);

	h_CurUploadMeshCount++;
	h_CurUploadVertexCount += vertexCount;
	h_CurUploadIndexCount += indexCount;

	return h_CurUploadMeshCount - 1;
}


__constant__ d_Color32 BGColor;
__constant__ int2 ScreenSize;
__constant__ int ScreenBufferSize;

__constant__ float4 WorldToViewMat[4];
__constant__ float4 WorldToNDCMat[4];
__constant__ float CamNear;
__constant__ float Inv_CamDis;

d_Color32* d_ColorBuffer;
float* d_DepthBuffer;
int h_ScreenSize;

Matrix4x4 h_WorldToViewMat;
Matrix4x4 h_WorldToNDCMat;

void CU_ChangeScreenSize(const ScreenPoint& screenSize)
{
	h_ScreenSize = screenSize.X * screenSize.Y;
	hipMemcpyToSymbol(HIP_SYMBOL(ScreenSize), &screenSize, sizeof(int2));
	hipMemcpyToSymbol(HIP_SYMBOL(ScreenBufferSize), &h_ScreenSize, sizeof(int));
}

void CU_BufferInit(Color32*& colorBuffer, float*& depthBuffer)
{
	hipMalloc(&d_ColorBuffer, sizeof(d_Color32) * h_ScreenSize);
	hipMalloc(&d_DepthBuffer, sizeof(float) * h_ScreenSize);
	hipHostMalloc(&depthBuffer, sizeof(float) * h_ScreenSize);
	hipHostRegister(colorBuffer, sizeof(d_Color32) * h_ScreenSize, hipHostRegisterMapped);
}
void CU_BufferFree(Color32*& colorBuffer, float*& depthBuffer)
{
	hipFree(d_ColorBuffer);
	hipFree(d_DepthBuffer);
	hipHostFree(depthBuffer);
	hipHostUnregister(colorBuffer);
	hipDeviceSynchronize();
}
__global__ void BG_Depth_Init(d_Color32* colorBuffer, float* depthBuffer)
{
	int idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx >= ScreenBufferSize)
	{
		return;
	}
	colorBuffer[idx] = BGColor;
	depthBuffer[idx] = 1.f;
}
void CU_BG_Depth_CamInfoSetting(const Color32& bgColor, const ScreenPoint& screenSize, Camera* pMainCam)
{
	hipMemcpyToSymbolAsync(HIP_SYMBOL(BGColor), &bgColor, sizeof(int), 0, hipMemcpyHostToDevice, S_BGDepthInit);
	dim3 block;
	int thread;
	Block_Thread_Size(h_ScreenSize, block, thread);
	BG_Depth_Init << <block, thread, 0, S_BGDepthInit >> > (d_ColorBuffer, d_DepthBuffer);

	h_WorldToNDCMat = pMainCam->GetWorldToNDCMat();
	hipMemcpyToSymbolAsync(HIP_SYMBOL(WorldToNDCMat), &h_WorldToNDCMat, sizeof(float4) * 4, 0, hipMemcpyHostToDevice);
	h_WorldToViewMat = pMainCam->GetWorldToViewMat();
	hipMemcpyToSymbolAsync(HIP_SYMBOL(WorldToViewMat), &h_WorldToViewMat, sizeof(float4) * 4, 0, hipMemcpyHostToDevice);

	float camNear, camFar, inv_CamDis;
	pMainCam->GetNearFar(camNear, camFar);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(CamNear), &camNear, sizeof(float), 0, hipMemcpyHostToDevice);
	inv_CamDis = 1 / (camFar - camNear);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(Inv_CamDis), &inv_CamDis, sizeof(float), 0, hipMemcpyHostToDevice);

}

void CU_SetRenderState()
{
	//���׸��� �߰� �� ������ ����
}

#pragma region DrawCall ó������

__constant__ int MaterialIdx;
__constant__ int GoCount;
int* d_MeshIdx;
int* h_MeshIdx;
d_Matrix4x4* d_LocalToNDCMat;
int h_GoCount;

#pragma region BoundCheckBox
//boundBox �ϴ� �ڽ��� ����
int* d_BCR;
float* d_log;
__global__ void BoundCheckBox(int* d_MeshIdx, int* d_BCR, d_Matrix4x4* d_LocalToNDCMat, float3* d_BvBox_Min, float3* d_BvBox_Max)
{
	int idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx >= GoCount)
	{
		return;
	}

	d_Matrix4x4 LocalToNDCMat = d_LocalToNDCMat[idx];
	float4 d_Frustum[6];
	float f12 = -LocalToNDCMat.Ele[12];
	float f13 = -LocalToNDCMat.Ele[13];
	float f14 = -LocalToNDCMat.Ele[14];
	float f15 = -LocalToNDCMat.Ele[15];

	//+y
	d_Frustum[0].x = f12 + LocalToNDCMat.Ele[4];
	d_Frustum[0].y = f13 + LocalToNDCMat.Ele[5];
	d_Frustum[0].z = f14 + LocalToNDCMat.Ele[6];
	d_Frustum[0].w = f15 + LocalToNDCMat.Ele[7];
	//-y
	d_Frustum[1].x = f12 - LocalToNDCMat.Ele[4];
	d_Frustum[1].y = f13 - LocalToNDCMat.Ele[5];
	d_Frustum[1].z = f14 - LocalToNDCMat.Ele[6];
	d_Frustum[1].w = f15 - LocalToNDCMat.Ele[7];
	//+x
	d_Frustum[2].x = f12 + LocalToNDCMat.Ele[0];
	d_Frustum[2].y = f13 + LocalToNDCMat.Ele[1];
	d_Frustum[2].z = f14 + LocalToNDCMat.Ele[2];
	d_Frustum[2].w = f15 + LocalToNDCMat.Ele[3];
	//-x
	d_Frustum[3].x = f12 - LocalToNDCMat.Ele[0];
	d_Frustum[3].y = f13 - LocalToNDCMat.Ele[1];
	d_Frustum[3].z = f14 - LocalToNDCMat.Ele[2];
	d_Frustum[3].w = f15 - LocalToNDCMat.Ele[3];
	//+z
	d_Frustum[4].x = f12 + LocalToNDCMat.Ele[8];
	d_Frustum[4].y = f13 + LocalToNDCMat.Ele[9];
	d_Frustum[4].z = f14 + LocalToNDCMat.Ele[10];
	d_Frustum[4].w = f15 + LocalToNDCMat.Ele[11];
	//-z
	d_Frustum[5].x = f12 - LocalToNDCMat.Ele[8];
	d_Frustum[5].y = f13 - LocalToNDCMat.Ele[9];
	d_Frustum[5].z = f14 - LocalToNDCMat.Ele[10];
	d_Frustum[5].w = f15 - LocalToNDCMat.Ele[11];

	float total_pow, inv_sqrt;

	total_pow = d_Frustum[0].x * d_Frustum[0].x + d_Frustum[0].y * d_Frustum[0].y + d_Frustum[0].z * d_Frustum[0].z;
	inv_sqrt = 1 / sqrtf(total_pow);
	d_Frustum[0].x *= inv_sqrt;
	d_Frustum[0].y *= inv_sqrt;
	d_Frustum[0].z *= inv_sqrt;
	d_Frustum[0].w *= inv_sqrt;

	total_pow = d_Frustum[1].x * d_Frustum[1].x + d_Frustum[1].y * d_Frustum[1].y + d_Frustum[1].z * d_Frustum[1].z;
	inv_sqrt = 1 / sqrtf(total_pow);
	d_Frustum[1].x *= inv_sqrt;
	d_Frustum[1].y *= inv_sqrt;
	d_Frustum[1].z *= inv_sqrt;
	d_Frustum[1].w *= inv_sqrt;

	total_pow = d_Frustum[2].x * d_Frustum[2].x + d_Frustum[2].y * d_Frustum[2].y + d_Frustum[2].z * d_Frustum[2].z;
	inv_sqrt = 1 / sqrtf(total_pow);
	d_Frustum[2].x *= inv_sqrt;
	d_Frustum[2].y *= inv_sqrt;
	d_Frustum[2].z *= inv_sqrt;
	d_Frustum[2].w *= inv_sqrt;

	total_pow = d_Frustum[3].x * d_Frustum[3].x + d_Frustum[3].y * d_Frustum[3].y + d_Frustum[3].z * d_Frustum[3].z;
	inv_sqrt = 1 / sqrtf(total_pow);
	d_Frustum[3].x *= inv_sqrt;
	d_Frustum[3].y *= inv_sqrt;
	d_Frustum[3].z *= inv_sqrt;
	d_Frustum[3].w *= inv_sqrt;

	total_pow = d_Frustum[4].x * d_Frustum[4].x + d_Frustum[4].y * d_Frustum[4].y + d_Frustum[4].z * d_Frustum[4].z;
	inv_sqrt = 1 / sqrtf(total_pow);
	d_Frustum[4].x *= inv_sqrt;
	d_Frustum[4].y *= inv_sqrt;
	d_Frustum[4].z *= inv_sqrt;
	d_Frustum[4].w *= inv_sqrt;

	total_pow = d_Frustum[5].x * d_Frustum[5].x + d_Frustum[5].y * d_Frustum[5].y + d_Frustum[5].z * d_Frustum[5].z;
	inv_sqrt = 1 / sqrtf(total_pow);
	d_Frustum[5].x *= inv_sqrt;
	d_Frustum[5].y *= inv_sqrt;
	d_Frustum[5].z *= inv_sqrt;
	d_Frustum[5].w *= inv_sqrt;

	int xDir, yDir, zDir, wDir; // 0 = ��� or 0 , 1 = ����
	float3 bv_Min = d_BvBox_Min[d_MeshIdx[idx]];
	float3 bv_Max = d_BvBox_Max[d_MeshIdx[idx]];

	float3 innerPos, outPos;
	float inDis, outDis;

	bool isIntersect = false;

	//inDis > 0 = outSide, outDis < 0 inSide
	//Inside = 1,
	//Intersect = 2,
	//Outside = 4

	float dirMulMin, dirMulMax;
	xDir = signbit(d_Frustum[0].x);
	yDir = signbit(d_Frustum[0].y);
	zDir = signbit(d_Frustum[0].z);
	wDir = signbit(d_Frustum[0].w);
	/*innerPos.x = bv_Min.x * (1 - xDir) + bv_Max.x * xDir;
	outPos.x = bv_Min.x * xDir + bv_Max.x * (1 - xDir);
	innerPos.y = bv_Min.y * (1 - yDir) + bv_Max.y * yDir;
	outPos.y = bv_Min.y * yDir + bv_Max.y * (1 - yDir);
	innerPos.z = bv_Min.z * (1 - zDir) + bv_Max.x * zDir;
	outPos.z = bv_Min.z * zDir + bv_Max.x * (1 - zDir);*/ //�ش� �� �ߺ��� ĳ���Ѱ� 1000�� ������ 0.5ms���� �������µ�
	dirMulMin = bv_Min.x * xDir;
	dirMulMax = bv_Max.x * xDir;
	innerPos.x = bv_Min.x - dirMulMin + dirMulMax;
	outPos.x = dirMulMin + bv_Max.x - dirMulMax;
	dirMulMin = bv_Min.y * yDir;
	dirMulMax = bv_Max.y * yDir;
	innerPos.y = bv_Min.y - dirMulMin + dirMulMax;
	outPos.y = dirMulMin + bv_Max.y - dirMulMax;
	dirMulMin = bv_Min.z * zDir;
	dirMulMax = bv_Max.z * zDir;
	innerPos.z = bv_Min.z - dirMulMin + dirMulMax;
	outPos.z = dirMulMin + bv_Max.z - dirMulMax;
	inDis = d_Frustum[0].x * innerPos.x + d_Frustum[0].y * innerPos.y + d_Frustum[0].z * innerPos.z + d_Frustum[0].w;
	outDis = d_Frustum[0].x * outPos.x + d_Frustum[0].y * outPos.y + d_Frustum[0].z * outPos.z + d_Frustum[0].w;
	if (inDis > 0)
	{
		d_BCR[idx] = 4;
		return;
	}
	else if (outDis >= 0)
	{
		isIntersect = true;
	}

	xDir = signbit(d_Frustum[1].x);
	yDir = signbit(d_Frustum[1].y);
	zDir = signbit(d_Frustum[1].z);
	wDir = signbit(d_Frustum[1].w);
	dirMulMin = bv_Min.x * xDir;
	dirMulMax = bv_Max.x * xDir;
	innerPos.x = bv_Min.x - dirMulMin + dirMulMax;
	outPos.x = dirMulMin + bv_Max.x - dirMulMax;
	dirMulMin = bv_Min.y * yDir;
	dirMulMax = bv_Max.y * yDir;
	innerPos.y = bv_Min.y - dirMulMin + dirMulMax;
	outPos.y = dirMulMin + bv_Max.y - dirMulMax;
	dirMulMin = bv_Min.z * zDir;
	dirMulMax = bv_Max.z * zDir;
	innerPos.z = bv_Min.z - dirMulMin + dirMulMax;
	outPos.z = dirMulMin + bv_Max.z - dirMulMax;
	inDis = d_Frustum[1].x * innerPos.x + d_Frustum[1].y * innerPos.y + d_Frustum[1].z * innerPos.z + d_Frustum[1].w;
	outDis = d_Frustum[1].x * outPos.x + d_Frustum[1].y * outPos.y + d_Frustum[1].z * outPos.z + d_Frustum[1].w;
	if (inDis > 0)
	{
		d_BCR[idx] = 4;
		return;
	}
	else if (outDis >= 0)
	{
		isIntersect = true;
	}

	xDir = signbit(d_Frustum[2].x);
	yDir = signbit(d_Frustum[2].y);
	zDir = signbit(d_Frustum[2].z);
	wDir = signbit(d_Frustum[2].w);
	dirMulMin = bv_Min.x * xDir;
	dirMulMax = bv_Max.x * xDir;
	innerPos.x = bv_Min.x - dirMulMin + dirMulMax;
	outPos.x = dirMulMin + bv_Max.x - dirMulMax;
	dirMulMin = bv_Min.y * yDir;
	dirMulMax = bv_Max.y * yDir;
	innerPos.y = bv_Min.y - dirMulMin + dirMulMax;
	outPos.y = dirMulMin + bv_Max.y - dirMulMax;
	dirMulMin = bv_Min.z * zDir;
	dirMulMax = bv_Max.z * zDir;
	innerPos.z = bv_Min.z - dirMulMin + dirMulMax;
	outPos.z = dirMulMin + bv_Max.z - dirMulMax;
	inDis = d_Frustum[2].x * innerPos.x + d_Frustum[2].y * innerPos.y + d_Frustum[2].z * innerPos.z + d_Frustum[2].w;
	outDis = d_Frustum[2].x * outPos.x + d_Frustum[2].y * outPos.y + d_Frustum[2].z * outPos.z + d_Frustum[2].w;
	if (inDis > 0)
	{
		d_BCR[idx] = 4;
		return;
	}
	else if (outDis >= 0)
	{
		isIntersect = true;
	}

	xDir = signbit(d_Frustum[3].x);
	yDir = signbit(d_Frustum[3].y);
	zDir = signbit(d_Frustum[3].z);
	wDir = signbit(d_Frustum[3].w);
	dirMulMin = bv_Min.x * xDir;
	dirMulMax = bv_Max.x * xDir;
	innerPos.x = bv_Min.x - dirMulMin + dirMulMax;
	outPos.x = dirMulMin + bv_Max.x - dirMulMax;
	dirMulMin = bv_Min.y * yDir;
	dirMulMax = bv_Max.y * yDir;
	innerPos.y = bv_Min.y - dirMulMin + dirMulMax;
	outPos.y = dirMulMin + bv_Max.y - dirMulMax;
	dirMulMin = bv_Min.z * zDir;
	dirMulMax = bv_Max.z * zDir;
	innerPos.z = bv_Min.z - dirMulMin + dirMulMax;
	outPos.z = dirMulMin + bv_Max.z - dirMulMax;
	inDis = d_Frustum[3].x * innerPos.x + d_Frustum[3].y * innerPos.y + d_Frustum[3].z * innerPos.z + d_Frustum[3].w;
	outDis = d_Frustum[3].x * outPos.x + d_Frustum[3].y * outPos.y + d_Frustum[3].z * outPos.z + d_Frustum[3].w;
	if (inDis > 0)
	{
		d_BCR[idx] = 4;
		return;
	}
	else if (outDis >= 0)
	{
		isIntersect = true;
	}

	xDir = signbit(d_Frustum[4].x);
	yDir = signbit(d_Frustum[4].y);
	zDir = signbit(d_Frustum[4].z);
	wDir = signbit(d_Frustum[4].w);
	dirMulMin = bv_Min.x * xDir;
	dirMulMax = bv_Max.x * xDir;
	innerPos.x = bv_Min.x - dirMulMin + dirMulMax;
	outPos.x = dirMulMin + bv_Max.x - dirMulMax;
	dirMulMin = bv_Min.y * yDir;
	dirMulMax = bv_Max.y * yDir;
	innerPos.y = bv_Min.y - dirMulMin + dirMulMax;
	outPos.y = dirMulMin + bv_Max.y - dirMulMax;
	dirMulMin = bv_Min.z * zDir;
	dirMulMax = bv_Max.z * zDir;
	innerPos.z = bv_Min.z - dirMulMin + dirMulMax;
	outPos.z = dirMulMin + bv_Max.z - dirMulMax;
	inDis = d_Frustum[4].x * innerPos.x + d_Frustum[4].y * innerPos.y + d_Frustum[4].z * innerPos.z + d_Frustum[4].w;
	outDis = d_Frustum[4].x * outPos.x + d_Frustum[4].y * outPos.y + d_Frustum[4].z * outPos.z + d_Frustum[4].w;
	if (inDis > 0)
	{
		d_BCR[idx] = 4;
		return;
	}
	else if (outDis >= 0)
	{
		isIntersect = true;
	}

	xDir = signbit(d_Frustum[5].x);
	yDir = signbit(d_Frustum[5].y);
	zDir = signbit(d_Frustum[5].z);
	wDir = signbit(d_Frustum[5].w);
	dirMulMin = bv_Min.x * xDir;
	dirMulMax = bv_Max.x * xDir;
	innerPos.x = bv_Min.x - dirMulMin + dirMulMax;
	outPos.x = dirMulMin + bv_Max.x - dirMulMax;
	dirMulMin = bv_Min.y * yDir;
	dirMulMax = bv_Max.y * yDir;
	innerPos.y = bv_Min.y - dirMulMin + dirMulMax;
	outPos.y = dirMulMin + bv_Max.y - dirMulMax;
	dirMulMin = bv_Min.z * zDir;
	dirMulMax = bv_Max.z * zDir;
	innerPos.z = bv_Min.z - dirMulMin + dirMulMax;
	outPos.z = dirMulMin + bv_Max.z - dirMulMax;
	inDis = d_Frustum[5].x * innerPos.x + d_Frustum[5].y * innerPos.y + d_Frustum[5].z * innerPos.z + d_Frustum[5].w;
	outDis = d_Frustum[5].x * outPos.x + d_Frustum[5].y * outPos.y + d_Frustum[5].z * outPos.z + d_Frustum[5].w;
	if (inDis > 0)
	{
		d_BCR[idx] = 4;
		return;
	}
	else if (outDis >= 0)
	{
		isIntersect = true;
	}

	if (isIntersect)
	{
		d_BCR[idx] = 2;
	}
	else
	{
		d_BCR[idx] = 1;
	}
}


int h_OutsideCount = 0;
int h_InsideCount = 0;
int h_IntersectCount = 0;
int h_DrawGoCount = 0;

int* h_BCR;
void CU_BoundCheckBox()
{
	dim3 block;
	int thread;
	Block_Thread_Size(h_GoCount, block, thread);
	BoundCheckBox << <block, thread >> > (d_MeshIdx, d_BCR, d_LocalToNDCMat, d_BvBox_Min, d_BvBox_Max);
	hipMemcpy(h_BCR, d_BCR, sizeof(int) * h_GoCount, hipMemcpyDeviceToHost);
}

#pragma endregion
#pragma region VertexSettingAndClipping
#pragma region functions
__device__ bool CheckisOut(int* pdirIdx, float4* pVt4, float* pSmallNumber)
{
	switch (*pdirIdx)
	{
	case 0:
		return pVt4->w < *pSmallNumber;
	case 1:
		return pVt4->z < -*pSmallNumber;
	case 2:
		return pVt4->z > 1 + *pSmallNumber;
	case 3:
		return pVt4->x > pVt4->w + *pSmallNumber;
	case 4:
		return pVt4->x < -pVt4->w - *pSmallNumber;
	case 5:
		return pVt4->y > pVt4->w + *pSmallNumber;
	case 6:
		return pVt4->y < -pVt4->w - *pSmallNumber;
	}
}
__device__ void FrustumClipping(int* pdirIdx, float4* pOutPos, float4* pInPos, float2* pOutUV, float2* pInUV, float4* pDesPos, float2* pDesUV)
{
	float t, s;
	switch (*pdirIdx)
	{
	case 0:
		t = pInPos->w / (pInPos->w - pOutPos->w);
		s = 1 - t;
		pDesPos->x = pOutPos->x * t + pInPos->x * s;
		pDesPos->y = pOutPos->y * t + pInPos->y * s;
		pDesPos->z = pOutPos->z * t + pInPos->z * s;
		pDesPos->w = 0;

		pDesUV->x = pOutUV->x * t + pInUV->x * s;
		pDesUV->y = pOutUV->y * t + pInUV->y * s;
		break;
	case 1:
		t = pInPos->z / (pInPos->z - pOutPos->z);
		s = 1 - t;
		pDesPos->x = pOutPos->x * t + pInPos->x * s;
		pDesPos->y = pOutPos->y * t + pInPos->y * s;
		pDesPos->w = pOutPos->w * t + pInPos->w * s;
		pDesPos->z = 0;

		pDesUV->x = pOutUV->x * t + pInUV->x * s;
		pDesUV->y = pOutUV->y * t + pInUV->y * s;
		break;
	case 2:
		t = (pInPos->z - 1) / (pInPos->z - pOutPos->z);
		s = 1 - t;
		pDesPos->x = pOutPos->x * t + pInPos->x * s;
		pDesPos->y = pOutPos->y * t + pInPos->y * s;
		pDesPos->w = pOutPos->w * t + pInPos->w * s;
		pDesPos->z = 1;

		pDesUV->x = pOutUV->x * t + pInUV->x * s;
		pDesUV->y = pOutUV->y * t + pInUV->y * s;
		break;
	case 3:
		t = (pInPos->w - pInPos->x) / (pInPos->w - pInPos->x - pOutPos->w + pOutPos->x);
		s = 1 - t;
		pDesPos->y = pOutPos->y * t + pInPos->y * s;
		pDesPos->z = pOutPos->z * t + pInPos->z * s;
		pDesPos->w = pOutPos->w * t + pInPos->w * s;
		pDesPos->x = pDesPos->w;

		pDesUV->x = pOutUV->x * t + pInUV->x * s;
		pDesUV->y = pOutUV->y * t + pInUV->y * s;
		break;
	case 4:
		t = (pInPos->w + pInPos->x) / (pInPos->w + pInPos->x - pOutPos->w - pOutPos->x);
		s = 1 - t;
		pDesPos->y = pOutPos->y * t + pInPos->y * s;
		pDesPos->z = pOutPos->z * t + pInPos->z * s;
		pDesPos->w = pOutPos->w * t + pInPos->w * s;
		pDesPos->x = -pDesPos->w;

		pDesUV->x = pOutUV->x * t + pInUV->x * s;
		pDesUV->y = pOutUV->y * t + pInUV->y * s;
		break;
	case 5:
		t = (pInPos->w - pInPos->y) / (pInPos->w - pInPos->y - pOutPos->w + pOutPos->y);
		s = 1 - t;
		pDesPos->x = pOutPos->x * t + pInPos->x * s;
		pDesPos->z = pOutPos->z * t + pInPos->z * s;
		pDesPos->w = pOutPos->w * t + pInPos->w * s;
		pDesPos->y = pDesPos->w;

		pDesUV->x = pOutUV->x * t + pInUV->x * s;
		pDesUV->y = pOutUV->y * t + pInUV->y * s;
		break;
	case 6:
		t = (pInPos->w + pInPos->y) / (pInPos->w + pInPos->y - pOutPos->w - pOutPos->y);
		s = 1 - t;
		pDesPos->x = pOutPos->x * t + pInPos->x * s;
		pDesPos->z = pOutPos->z * t + pInPos->z * s;
		pDesPos->w = pOutPos->w * t + pInPos->w * s;
		pDesPos->y = -pDesPos->w;

		pDesUV->x = pOutUV->x * t + pInUV->x * s;
		pDesUV->y = pOutUV->y * t + pInUV->y * s;
		break;
	}
}
#pragma endregion
float4* d_VertexPos;
float2* d_VertexUV;
bool* d_isDraw;
int* d_DrawTriangleIdxPrefixSum;
bool* h_isIntersect;
bool* d_isIntersect;
int* h_TriangleIdx;
int* d_TriangleIdx;
int* h_TriangleGoIdx;
int* d_TriangleGoIdx;
int* h_VertexStartIdx;
int* d_VertexStartIdx;

int h_TriangleCount;
__constant__ int TriangleCount;
int h_VertexCountAfterClip;
__constant__ int VertexCountAfterClip;
int h_TriangleCountAfterClip;
__constant__ int TriangleCountAfterClip;


__global__ void VertexSetting(float4* d_VertexPos, float2* d_VertexUV, bool* d_isDraw, bool* d_isIntersect, int* d_TriangleIdx, int* d_TriangleGoIdx, int* d_VertexStartIdx, d_Matrix4x4* d_LocalToNDCMat,
	int* d_MeshIdx, int* d_MeshIndexStartIdx, int* d_MeshIndices, int* d_MeshVertexStartIdx, float3* d_MeshVertices, float2* d_MeshUVs)
{
	int idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx >= TriangleCount)
	{
		return;
	}

	//return; // 6~7

	int triangleIdx = d_TriangleIdx[idx];
	int goIdx = d_TriangleGoIdx[idx];
	int meshIdx = d_MeshIdx[goIdx]; //�޽��� ��ȣ

	int meshIndex; //�޽� �ȿ����� �ε���
	int vStartIdx = d_VertexStartIdx[idx];
	int vIdx;
	float3 meshVertex;
	float inv_w;
	float4 VertexPos;
	d_Matrix4x4 localToNDCMat = d_LocalToNDCMat[goIdx];
	int curVertCount = 3;
	int curTriangleCount = curVertCount - 2;

	vIdx = vStartIdx + 0;
	meshIndex = d_MeshIndices[d_MeshIndexStartIdx[meshIdx] + triangleIdx * 3 + 0];
	meshVertex = d_MeshVertices[d_MeshVertexStartIdx[meshIdx] + meshIndex];
	d_VertexUV[vIdx] = d_MeshUVs[d_MeshVertexStartIdx[meshIdx] + meshIndex];
	VertexPos.x = localToNDCMat.Ele[0] * meshVertex.x + localToNDCMat.Ele[1] * meshVertex.y + localToNDCMat.Ele[2] * meshVertex.z + localToNDCMat.Ele[3];
	VertexPos.y = localToNDCMat.Ele[4] * meshVertex.x + localToNDCMat.Ele[5] * meshVertex.y + localToNDCMat.Ele[6] * meshVertex.z + localToNDCMat.Ele[7];
	VertexPos.w = localToNDCMat.Ele[12] * meshVertex.x + localToNDCMat.Ele[13] * meshVertex.y + localToNDCMat.Ele[14] * meshVertex.z + localToNDCMat.Ele[15];
	VertexPos.z = ((VertexPos.w - CamNear) * Inv_CamDis);
	d_VertexPos[vIdx] = VertexPos;

	vIdx = vStartIdx + 1;
	meshIndex = d_MeshIndices[d_MeshIndexStartIdx[meshIdx] + triangleIdx * 3 + 1];
	meshVertex = d_MeshVertices[d_MeshVertexStartIdx[meshIdx] + meshIndex];
	d_VertexUV[vIdx] = d_MeshUVs[d_MeshVertexStartIdx[meshIdx] + meshIndex];
	VertexPos.x = localToNDCMat.Ele[0] * meshVertex.x + localToNDCMat.Ele[1] * meshVertex.y + localToNDCMat.Ele[2] * meshVertex.z + localToNDCMat.Ele[3];
	VertexPos.y = localToNDCMat.Ele[4] * meshVertex.x + localToNDCMat.Ele[5] * meshVertex.y + localToNDCMat.Ele[6] * meshVertex.z + localToNDCMat.Ele[7];
	VertexPos.w = localToNDCMat.Ele[12] * meshVertex.x + localToNDCMat.Ele[13] * meshVertex.y + localToNDCMat.Ele[14] * meshVertex.z + localToNDCMat.Ele[15];
	VertexPos.z = ((VertexPos.w - CamNear) * Inv_CamDis);
	d_VertexPos[vIdx] = VertexPos;

	vIdx = vStartIdx + 2;
	meshIndex = d_MeshIndices[d_MeshIndexStartIdx[meshIdx] + triangleIdx * 3 + 2];
	meshVertex = d_MeshVertices[d_MeshVertexStartIdx[meshIdx] + meshIndex];
	d_VertexUV[vIdx] = d_MeshUVs[d_MeshVertexStartIdx[meshIdx] + meshIndex];
	VertexPos.x = localToNDCMat.Ele[0] * meshVertex.x + localToNDCMat.Ele[1] * meshVertex.y + localToNDCMat.Ele[2] * meshVertex.z + localToNDCMat.Ele[3];
	VertexPos.y = localToNDCMat.Ele[4] * meshVertex.x + localToNDCMat.Ele[5] * meshVertex.y + localToNDCMat.Ele[6] * meshVertex.z + localToNDCMat.Ele[7];
	VertexPos.w = localToNDCMat.Ele[12] * meshVertex.x + localToNDCMat.Ele[13] * meshVertex.y + localToNDCMat.Ele[14] * meshVertex.z + localToNDCMat.Ele[15];
	VertexPos.z = ((VertexPos.w - CamNear) * Inv_CamDis);
	d_VertexPos[vIdx] = VertexPos;

	//return; // 8~9

	if (d_isIntersect[idx])
	{
		//clipping ó��
		int dirIdx;
		int prev, cur, next;
		bool arr_isOut[9];
		float4 arr_Pos_ClipBefore[9];
		float4 arr_Pos_ClipResult[9];
		float2 arr_UV_ClipBefore[9];
		float2 arr_UV_ClipResult[9];
		float small_Number = 1.e-6f;
		int writeIdx = 0;

		arr_Pos_ClipBefore[0] = d_VertexPos[vStartIdx];
		arr_Pos_ClipBefore[1] = d_VertexPos[vStartIdx + 1];
		arr_Pos_ClipBefore[2] = d_VertexPos[vStartIdx + 2];
		arr_UV_ClipBefore[0] = d_VertexUV[vStartIdx];
		arr_UV_ClipBefore[1] = d_VertexUV[vStartIdx + 1];
		arr_UV_ClipBefore[2] = d_VertexUV[vStartIdx + 2];
		//return; // 9~10
#pragma region forloop
		for (int dirIdx = 0; dirIdx < 7; dirIdx++)
		{
			for (int i = 0; i < curVertCount; i++)
			{
				arr_isOut[i] = CheckisOut(&dirIdx, &arr_Pos_ClipBefore[i], &small_Number);
			}

			for (int i = 0; i < curVertCount; i++)
			{
				//���ϵǴ� �б�� �������
				if (i == 0)
				{
					prev = curVertCount - 2;
					cur = curVertCount - 1;
					next = 0;
				}
				else if (i == 1)
				{
					prev = curVertCount - 1;
					cur = 0;
					next = 1;
				}
				else
				{
					prev = i - 2;
					cur = i - 1;
					next = i;
				}
				if (arr_isOut[cur] == true)
				{
					if (arr_isOut[prev] == false)
					{
						FrustumClipping(&dirIdx, &arr_Pos_ClipBefore[cur], &arr_Pos_ClipBefore[prev], &arr_UV_ClipBefore[cur], &arr_UV_ClipBefore[prev], &arr_Pos_ClipResult[writeIdx], &arr_UV_ClipResult[writeIdx]);
						writeIdx++;
					}
					if (arr_isOut[next] == false)
					{
						FrustumClipping(&dirIdx, &arr_Pos_ClipBefore[cur], &arr_Pos_ClipBefore[next], &arr_UV_ClipBefore[cur], &arr_UV_ClipBefore[next], &arr_Pos_ClipResult[writeIdx], &arr_UV_ClipResult[writeIdx]);
						writeIdx++;
					}
				}
				else
				{
					//�״�� ����
					arr_Pos_ClipResult[writeIdx] = arr_Pos_ClipBefore[cur];
					arr_UV_ClipResult[writeIdx] = arr_UV_ClipBefore[cur];
					writeIdx++;
				}
			}
			curVertCount = writeIdx;
			writeIdx = 0;
			for (int i = 0; i < curVertCount; i++)
			{
				arr_isOut[i] = false;
				arr_Pos_ClipBefore[i] = arr_Pos_ClipResult[i];
				arr_UV_ClipBefore[i] = arr_UV_ClipResult[i];
			}
		}

#pragma endregion

		//���� forloop�� ������ �ʱ�ȭ�� ����
		//curVertCount = ���� Ŭ���� �� �� ����
		//arr_Pos_ClipResult�� ��� �������
		curTriangleCount = curVertCount - 2;
		for (int i = 0; i < curTriangleCount; i++)
		{
			vIdx = vStartIdx + i * 3;
			d_VertexPos[vIdx] = arr_Pos_ClipResult[0];
			d_VertexPos[vIdx + 1] = arr_Pos_ClipResult[i + 1];
			d_VertexPos[vIdx + 2] = arr_Pos_ClipResult[i + 2];
			d_VertexUV[vIdx] = arr_UV_ClipResult[0];
			d_VertexUV[vIdx + 1] = arr_UV_ClipResult[i + 1];
			d_VertexUV[vIdx + 2] = arr_UV_ClipResult[i + 2];
		}
	}
	int value = vStartIdx / 3;
	for (int i = 0; i < curTriangleCount; i++)
	{
		d_isDraw[value + i] = true;

		vIdx = vStartIdx + i * 3;
		inv_w = 1 / d_VertexPos[vIdx].w;
		d_VertexPos[vIdx].x *= inv_w;
		d_VertexPos[vIdx].y *= inv_w;
		inv_w = 1 / d_VertexPos[vIdx + 1].w;
		d_VertexPos[vIdx + 1].x *= inv_w;
		d_VertexPos[vIdx + 1].y *= inv_w;
		inv_w = 1 / d_VertexPos[vIdx + 2].w;
		d_VertexPos[vIdx + 2].x *= inv_w;
		d_VertexPos[vIdx + 2].y *= inv_w;
	}
}

void CU_VertexArraySetting()
{
	hipStream_t S_VertexArraySetting;
	hipStreamCreate(&S_VertexArraySetting);
	int insideCount = 0;
	int intersectCount = 0;
	int outsideCount = 0;
	int insideTriangleCount = 0;
	int intersectTriangleCount = 0;
	for (int i = 0; i < h_GoCount; i++)
	{
		switch (h_BCR[i])
		{
		case  1: //inside
			insideTriangleCount += h_MeshTriangleCountLookUpTable[h_MeshIdx[i]];
			insideCount++;
			break;
		case 2: //intersect
			intersectTriangleCount += h_MeshTriangleCountLookUpTable[h_MeshIdx[i]];
			intersectCount++;
			break;
		case 4: //outside
			break;
		}
	}

	h_TriangleCount = insideTriangleCount + intersectTriangleCount;
	h_VertexCountAfterClip = insideTriangleCount * 3 + intersectTriangleCount * 21;
	h_TriangleCountAfterClip = h_VertexCountAfterClip / 3;

	//intersectTriangleCount �� ����clipping �� �ִ� 6������ �߻����� �׷� �ﰢ���� 7���̹Ƿ� ���� 21��
	static int TriangleArrayCount = 0;
	if (TriangleArrayCount < h_TriangleCount)
	{
		if (TriangleArrayCount != 0)
		{
			delete[] h_isIntersect;
			hipFree(d_isIntersect);
			delete[] h_TriangleIdx;
			hipFree(d_TriangleIdx);
			delete[] h_TriangleGoIdx;
			hipFree(d_TriangleGoIdx);
			delete[] h_VertexStartIdx;
			hipFree(d_VertexStartIdx);
		}
		TriangleArrayCount = h_TriangleCount + (h_TriangleCount >> 2);
		h_isIntersect = new bool[TriangleArrayCount];
		hipMalloc(&d_isIntersect, sizeof(bool) * TriangleArrayCount);
		h_TriangleIdx = new int[TriangleArrayCount];
		hipMalloc(&d_TriangleIdx, sizeof(int) * TriangleArrayCount);
		h_TriangleGoIdx = new int[TriangleArrayCount];
		hipMalloc(&d_TriangleGoIdx, sizeof(int) * TriangleArrayCount);
		h_VertexStartIdx = new int[TriangleArrayCount];
		hipMalloc(&d_VertexStartIdx, sizeof(int) * TriangleArrayCount);
	}

	static int VertexArrayCountAfterClip = 0;
	static int TriangleArrayCountAfterClip = 0;
	if (VertexArrayCountAfterClip < h_VertexCountAfterClip)
	{
		if (VertexArrayCountAfterClip != 0)
		{
			hipFree(d_VertexPos);
			hipFree(d_VertexUV);
			hipFree(d_isDraw);
			hipFree(d_DrawTriangleIdxPrefixSum);
		}
		VertexArrayCountAfterClip = h_VertexCountAfterClip + (h_VertexCountAfterClip >> 2);
		TriangleArrayCountAfterClip = VertexArrayCountAfterClip / 3;
		hipMalloc(&d_isDraw, sizeof(bool) * TriangleArrayCountAfterClip);
		hipMalloc(&d_DrawTriangleIdxPrefixSum, sizeof(int) * TriangleArrayCountAfterClip);
		hipMalloc(&d_VertexPos, sizeof(float4) * VertexArrayCountAfterClip);
		hipMalloc(&d_VertexUV, sizeof(float4) * VertexArrayCountAfterClip);
	}

	hipMemcpyToSymbolAsync(HIP_SYMBOL(TriangleCount), &h_TriangleCount, sizeof(int), 0, hipMemcpyHostToDevice, S_VertexArraySetting);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(VertexCountAfterClip), &h_VertexCountAfterClip, sizeof(int), 0, hipMemcpyHostToDevice, S_VertexArraySetting);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(TriangleCountAfterClip), &h_TriangleCountAfterClip, sizeof(int), 0, hipMemcpyHostToDevice, S_VertexArraySetting);
	hipMemsetAsync(d_isDraw, 0, sizeof(bool) * h_VertexCountAfterClip / 3, S_VertexArraySetting);

	int curMeshTriangleCount = 0;
	int curTriangleIdx = 0;
	int curVertexIdx = 0;

	for (int i = 0; i < h_GoCount; i++)
	{
		switch (h_BCR[i])
		{
		case 1: //inside
			curMeshTriangleCount = h_MeshTriangleCountLookUpTable[h_MeshIdx[i]];
			for (int j = 0; j < curMeshTriangleCount; j++)
			{
				h_TriangleIdx[curTriangleIdx] = j;
				h_TriangleGoIdx[curTriangleIdx] = i;
				h_isIntersect[curTriangleIdx] = false;
				h_VertexStartIdx[curTriangleIdx] = curVertexIdx;
				curTriangleIdx++;
				curVertexIdx += 3;
			}
			break;
		case 2: //intersect
			curMeshTriangleCount = h_MeshTriangleCountLookUpTable[h_MeshIdx[i]];
			for (int j = 0; j < curMeshTriangleCount; j++)
			{
				h_TriangleIdx[curTriangleIdx] = j;
				h_TriangleGoIdx[curTriangleIdx] = i;
				h_isIntersect[curTriangleIdx] = true;
				h_VertexStartIdx[curTriangleIdx] = curVertexIdx;
				curTriangleIdx++;
				curVertexIdx += 21;
			}
			break;
		}
	}
	hipMemcpyAsync(d_TriangleIdx, h_TriangleIdx, sizeof(int) * h_TriangleCount, hipMemcpyHostToDevice, S_VertexArraySetting);
	hipMemcpyAsync(d_TriangleGoIdx, h_TriangleGoIdx, sizeof(int) * h_TriangleCount, hipMemcpyHostToDevice, S_VertexArraySetting);
	hipMemcpyAsync(d_isIntersect, h_isIntersect, sizeof(bool) * h_TriangleCount, hipMemcpyHostToDevice, S_VertexArraySetting);
	hipMemcpyAsync(d_VertexStartIdx, h_VertexStartIdx, sizeof(int) * h_TriangleCount, hipMemcpyHostToDevice, S_VertexArraySetting);

	dim3 block;
	int thread;
	Block_Thread_Size(h_TriangleCount, block, thread);

	VertexSetting << <block, thread, 0, S_VertexArraySetting >> > (d_VertexPos, d_VertexUV, d_isDraw, d_isIntersect, d_TriangleIdx, d_TriangleGoIdx, d_VertexStartIdx, d_LocalToNDCMat,
		d_MeshIdx, d_MeshIndexStartIdx, d_MeshIndices, d_MeshVertexStartIdx, d_MeshVertices, d_MeshUVs);
	hipStreamSynchronize(S_VertexArraySetting);
	hipStreamDestroy(S_VertexArraySetting);
}
#pragma endregion

__global__ void Culling(float4* d_VertexPos, bool* d_isDraw, int* d_DrawTriangleIdxPrefixSum)
{
	extern __shared__ int shared[];
	int idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx >= TriangleCountAfterClip)
	{
		return;
	}

	if (d_isDraw[idx] == true)
	{
		float4 pos0 = d_VertexPos[idx * 3];
		float4 pos1 = d_VertexPos[idx * 3 + 1];
		float4 pos2 = d_VertexPos[idx * 3 + 2];

		float x_12, y_12, x_13, y_13;
		x_12 = pos1.x - pos0.x;
		y_12 = pos1.y - pos0.y;
		x_13 = pos2.x - pos0.x;
		y_13 = pos2.y - pos0.y;
		d_isDraw[idx] = x_12 * y_13 - y_12 * x_13 < 0;
	}

	shared[threadIdx.x] = d_isDraw[idx];
	for (int stride = 1; stride < blockDim.x; stride <<= 1)
	{
		__syncthreads();
		if (threadIdx.x >= stride)
		{
			shared[threadIdx.x] += shared[threadIdx.x - stride];
		}
	}
	d_DrawTriangleIdxPrefixSum[idx] = shared[threadIdx.x];
}
__global__ void SetDrawTriangleIdxPrefixCom(bool* d_isDraw, int* d_DrawTriangleIdxPrefixSum, int* d_DrawTriangleIdxPrefixSumCom)
{
	int idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx >= TriangleCountAfterClip)
	{
		return;
	}
	if (d_isDraw[idx] == true)
	{
		int comIdx = d_DrawTriangleIdxPrefixSum[idx] - 1;
		d_DrawTriangleIdxPrefixSumCom[comIdx] = idx;
	}
}

int* d_DrawTriangleIdxPrefixSumCom;
float4* d_DrawVertexPos;
float2* d_DrawVertexUV;

__constant__ int DrawTriangleCount;
int h_DrawTriangleCount;
int* d_VerticalBottom;
int* d_VerticalHeightPrefixSum;

int h_DrawTriangleHeightCount;
__constant__ int DrawTriangleHeightCount;
int* d_DrawTriangleHeightTriangleIdx;
int* d_DrawTriangleHeight;
int* d_DrawTriangleHeightLeft;
int* d_DrawTriangleHeightRight;
float2* d_DTHLeftUV;
float2* d_DTHRightUV;
float* d_DTHLeftDepth;
float* d_DTHRightDepth;
int* d_FragmentCountPrefixSum;

int h_FragmentCount;
__constant__ int FragmentCount;
int* d_FragmentDTHIdx; //DTH = DrawTriangleHeight


__global__ void DrawVertexSetting_GetVerticalRange(float4* d_VertexPos, float2* d_VertexUV, float4* d_DrawVertexPos, float2* d_DrawVertexUV, int* d_DrawTriangleIdxPrefixSumCom, int* d_VerticalBottom, int* d_VerticalHeightPrefixSum)
{
	extern __shared__ int shared[];
	int idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx >= DrawTriangleCount) return;

	int vIdx0 = d_DrawTriangleIdxPrefixSumCom[idx] * 3;
	int vIdx1 = vIdx0 + 1;
	int vIdx2 = vIdx0 + 2;
	int DrawvIdx0 = idx * 3;
	int DrawvIdx1 = DrawvIdx0 + 1;
	int DrawvIdx2 = DrawvIdx0 + 2;

	float4 vPos0 = d_VertexPos[vIdx0];
	float4 vPos1 = d_VertexPos[vIdx1];
	float4 vPos2 = d_VertexPos[vIdx2];

	float halfScreenX = ScreenSize.x * 0.5f;
	float halfScreenY = ScreenSize.y * 0.5f;

	vPos0.x = vPos0.x * halfScreenX + halfScreenX;
	vPos1.x = vPos1.x * halfScreenX + halfScreenX;
	vPos2.x = vPos2.x * halfScreenX + halfScreenX;
	vPos0.y = -(vPos0.y * halfScreenY) + halfScreenY;
	vPos1.y = -(vPos1.y * halfScreenY) + halfScreenY;
	vPos2.y = -(vPos2.y * halfScreenY) + halfScreenY;

	d_DrawVertexUV[DrawvIdx0] = d_VertexUV[vIdx0];
	d_DrawVertexUV[DrawvIdx1] = d_VertexUV[vIdx1];
	d_DrawVertexUV[DrawvIdx2] = d_VertexUV[vIdx2];
	d_DrawVertexPos[DrawvIdx0] = vPos0;
	d_DrawVertexPos[DrawvIdx1] = vPos1;
	d_DrawVertexPos[DrawvIdx2] = vPos2;

	float top = max(max(vPos0.y, vPos1.y), vPos2.y);
	float bottom = min(min(vPos0.y, vPos1.y), vPos2.y);

	int iTop, iBottom;
	int iValue = (int)bottom;
	int addValue = bottom > iValue + 0.5f ? 1 : 0;
	iBottom = iValue + addValue;

	iValue = (int)top;
	addValue = top > iValue + 0.5f ? 0 : -1;
	iTop = iValue + addValue;

	d_VerticalBottom[idx] = iBottom;
	shared[threadIdx.x] = iTop - iBottom + 1;
	for (int stride = 1; stride < blockDim.x; stride <<= 1)
	{
		__syncthreads();
		if (threadIdx.x >= stride)
		{
			shared[threadIdx.x] += shared[threadIdx.x - stride];
		}
	}

	d_VerticalHeightPrefixSum[idx] = shared[threadIdx.x];
}
__global__ void SetDrawTriangleHeightTriangleIdx(int* d_VerticalHeightPrefixSum, int* d_DrawTriangleHeightTriangleIdx)
{
	int idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx >= DrawTriangleCount)	return;

	int start = idx == 0 ? 0 : d_VerticalHeightPrefixSum[idx - 1];
	int end = d_VerticalHeightPrefixSum[idx];
	for (int i = start; i < end; i++)
	{
		d_DrawTriangleHeightTriangleIdx[i] = idx;
	}
}
__global__ void SetDrawTriangleHeight(int* d_VerticalBottom, int* d_VerticalHeightPrefixSum, int* d_DrawTriangleHeightTriangleIdx, int* d_DrawTriangleHeight)
{
	int idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx >= DrawTriangleHeightCount)	return;

	int triangleIdx = d_DrawTriangleHeightTriangleIdx[idx];
	int beforeSum = triangleIdx == 0 ? 0 : d_VerticalHeightPrefixSum[triangleIdx - 1];

	d_DrawTriangleHeight[idx] = d_VerticalBottom[triangleIdx] + idx - beforeSum;
}
__global__ void SetDrawTriangleHeightInfo(float4* d_DrawVertexPos, float2* d_DrawVertexUV, int* d_DrawTriangleHeightTriangleIdx, int* d_DrawTriangleHeight, 
	int* d_DrawTriangleHeightLeft, int* d_DrawTriangleHeightRight, float2* d_DTHLeftUV, float2* d_DTHRightUV, float* d_DTHLeftDepth, float* d_DTHRightDepth, 
	int* d_FragmentCountPrefixSum, float* d_Log)
{
	extern __shared__ int shared[];

	int idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx >= DrawTriangleHeightCount)	return;

	int triangleIdx = d_DrawTriangleHeightTriangleIdx[idx];
	int iHeight = d_DrawTriangleHeight[idx];
	float fHeight = iHeight + 0.5f;

	int vIdx = triangleIdx * 3;
	float4 va = d_DrawVertexPos[vIdx];
	float4 vb = d_DrawVertexPos[vIdx + 1];
	float4 vc = d_DrawVertexPos[vIdx + 2];
	float4 firstv, secondv;
	float2 a, b;
	a.x = vc.x - va.x;
	a.y = vc.y - va.y;
	b.x = vc.x - vb.x;
	b.y = vc.y - vb.y;
	float aDota = a.x * a.x + a.y * a.y;
	float bDotb = b.x * b.x + b.y * b.y;
	float aDotb = a.x * b.x + a.y * b.y;
	float inv_a_w = 1 / va.w;
	float inv_b_w = 1 / vb.w;

	float2 pos, d, leftUV, rightUV;
	float leftDepth, rightDepth;
	float dDota, dDotb, denominator, inv_denominator, s, t, oneMinusST, inv_wa_t, inv_wb_s, inv_wc_oMST, inv_w_total;

	float2 uva = d_DrawVertexUV[vIdx];
	float2 uvb = d_DrawVertexUV[vIdx + 1];
	float2 uvc = d_DrawVertexUV[vIdx + 2];

	float horvalue[2];
	int horvalueIdx = 0;
	if (va.y != vb.y)
	{
		if (va.y < vb.y)
		{
			firstv = va;
			secondv = vb;
		}
		else
		{
			firstv = vb;
			secondv = va;
		}
		if (firstv.y <= fHeight && secondv.y >= fHeight)
		{
			float t = (fHeight - secondv.y) / (firstv.y - secondv.y);
			horvalue[horvalueIdx] = firstv.x * t + secondv.x * (1 - t);
			horvalueIdx++;
		}
	}
	if (vb.y != vc.y)
	{
		if (vb.y < vc.y)
		{
			firstv = vb;
			secondv = vc;
		}
		else
		{
			firstv = vc;
			secondv = vb;
		}
		if (firstv.y <= fHeight && secondv.y >= fHeight)
		{
			float t = (fHeight - secondv.y) / (firstv.y - secondv.y);
			horvalue[horvalueIdx] = firstv.x * t + secondv.x * (1 - t);
			horvalueIdx++;
		}
	}

	if (vc.y != va.y)
	{
		if (vc.y < va.y)
		{
			firstv = vc;
			secondv = va;
		}
		else
		{
			firstv = va;
			secondv = vc;
		}
		if (firstv.y <= fHeight && secondv.y >= fHeight)
		{
			float t = (fHeight - secondv.y) / (firstv.y - secondv.y);
			horvalue[horvalueIdx] = firstv.x * t + secondv.x * (1 - t);
			horvalueIdx++;
		}
	}

	float fRight = max(horvalue[0], horvalue[1]);
	float fLeft = min(horvalue[0], horvalue[1]);
	int iRight, iLeft;

	int iValue = (int)fLeft;
	int addValue = fLeft > iValue + 0.5f ? 1 : 0;
	iLeft = iValue + addValue;
	d_DrawTriangleHeightLeft[idx] = iLeft;

	iValue = (int)fRight;
	addValue = fRight > iValue + 0.5f ? 0 : -1;
	iRight = iValue + addValue;
	d_DrawTriangleHeightRight[idx] = iRight;

	int width = iRight - iLeft + 1;

	pos.x = fLeft;
	pos.y = fHeight;

	d.x = vc.x - pos.x;
	d.y = vc.y - pos.y;

	dDota = d.x * a.x + d.y * a.y;
	dDotb = d.x * b.x + d.y * b.y;
	denominator = aDotb * aDotb - aDota * bDotb;

	inv_denominator = 1 / denominator;
	s = (dDota * aDotb - dDotb * aDota) * inv_denominator;
	t = (dDotb * aDotb - dDota * bDotb) * inv_denominator;
	oneMinusST = 1 - s - t;

	leftDepth = va.z * t + vb.z * s + vc.z * oneMinusST;
	d_DTHLeftDepth[idx] = leftDepth;

	inv_wa_t = t * inv_a_w;
	inv_wb_s = s * inv_b_w;
	inv_wc_oMST = oneMinusST / vc.w;

	inv_w_total = 1 / (inv_wa_t + inv_wb_s + inv_wc_oMST);
	leftUV.x = (uva.x * inv_wa_t + uvb.x * inv_wb_s + uvc.x * inv_wc_oMST) * inv_w_total;
	leftUV.y = (uva.y * inv_wa_t + uvb.y * inv_wb_s + uvc.y * inv_wc_oMST) * inv_w_total;
	d_DTHLeftUV[idx] = leftUV;


	pos.x = fRight;

	d.x = vc.x - pos.x;

	dDota = d.x * a.x + d.y * a.y;
	dDotb = d.x * b.x + d.y * b.y;
	denominator = aDotb * aDotb - aDota * bDotb;

	inv_denominator = 1 / denominator;
	s = (dDota * aDotb - dDotb * aDota) * inv_denominator;
	t = (dDotb * aDotb - dDota * bDotb) * inv_denominator;
	oneMinusST = 1 - s - t;

	rightDepth = va.z * t + vb.z * s + vc.z * oneMinusST;
	d_DTHRightDepth[idx] = rightDepth;

	inv_wa_t = t * inv_a_w;
	inv_wb_s = s * inv_b_w;
	inv_wc_oMST = oneMinusST / vc.w;

	inv_w_total = 1 / (inv_wa_t + inv_wb_s + inv_wc_oMST);
	rightUV.x = (uva.x * inv_wa_t + uvb.x * inv_wb_s + uvc.x * inv_wc_oMST) * inv_w_total;
	rightUV.y = (uva.y * inv_wa_t + uvb.y * inv_wb_s + uvc.y * inv_wc_oMST) * inv_w_total;
	d_DTHRightUV[idx] = rightUV;
	shared[threadIdx.x] = width;
	for (int stride = 1; stride < blockDim.x; stride <<= 1)
	{
		__syncthreads();
		if (threadIdx.x >= stride)
		{
			shared[threadIdx.x] += shared[threadIdx.x - stride];
		}
	}

	d_FragmentCountPrefixSum[idx] = shared[threadIdx.x];
}
__global__ void SetFragmentDTHIdx(int* d_FragmentCountPrefixSum, int* d_FragmentDTHIdx)
{
	int idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx >= DrawTriangleHeightCount)	return;

	int start = idx == 0 ? 0 : d_FragmentCountPrefixSum[idx - 1];
	int end = d_FragmentCountPrefixSum[idx];
	for (int i = start; i < end; i++)
	{
		d_FragmentDTHIdx[i] = idx;
	}
}
__global__ void SetBuffer(int* d_FragmentDTHIdx, int* d_FragmentCountPrefixSum, int* d_DrawTriangleHeight, int* d_DrawTriangleHeightLeft, int* d_DrawTriangleHeightRight, float2* d_DTHLeftUV, float2* d_DTHRightUV, float* d_DTHLeftDepth, float* d_DTHRightDepth,
	d_Color32* d_TexBuffer, int* d_TexWidth, int* d_TexHeight, d_Color32* d_ColorBuffer, float* d_DepthBuffer, float* d_Log)
{
	//return; // 45
	int idx = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
	if (idx >= FragmentCount)	return;

	int DTHIdx = d_FragmentDTHIdx[idx];
	int beforePrefixSum = DTHIdx == 0 ? 0 : d_FragmentCountPrefixSum[DTHIdx - 1];
	int tex_xIdx, tex_yIdx, texIdx, bufferIdx;

	int texWidth = d_TexWidth[0];
	int texHeight = d_TexHeight[0];

	int iLeft = d_DrawTriangleHeightLeft[DTHIdx];
	int iRight = d_DrawTriangleHeightRight[DTHIdx];
	float2 leftUV = d_DTHLeftUV[DTHIdx];
	float2 rightUV = d_DTHRightUV[DTHIdx];
	float leftDepth = d_DTHLeftDepth[DTHIdx];
	float rightDepth = d_DTHRightDepth[DTHIdx];
	int iHeight = d_DrawTriangleHeight[DTHIdx];
	int iX = iLeft + idx - beforePrefixSum;

	int n = iRight - iLeft;
	int k = iX - iLeft;
	float ratio = (k * leftDepth) / (k * leftDepth + (n - k) * rightDepth);
	float curDepth = leftDepth + (rightDepth - leftDepth) * ratio;
	bufferIdx = ScreenSize.x * iHeight + iX;
	
	atomicMin(&d_DepthBuffer[bufferIdx], curDepth);

	if (d_DepthBuffer[bufferIdx] == curDepth)
	{
		float2 curUV;
		curUV.x = leftUV.x + (rightUV.x - leftUV.x) * ratio;
		curUV.y = leftUV.y + (rightUV.y - leftUV.y) * ratio;
		tex_xIdx = max(0, min((int)(curUV.x * texWidth), texWidth - 1));
		tex_yIdx = max(0, min((int)(curUV.y * texHeight), texHeight - 1));
		texIdx = tex_xIdx + tex_yIdx * texWidth;
		d_ColorBuffer[bufferIdx] = d_TexBuffer[texIdx];
	}
	

	//return; // 124 ~ 127
}


void CU_Culling_ScanLine_SetBuffer()
{
	hipStream_t S_CullingScanLineRange;
	hipStreamCreate(&S_CullingScanLineRange);
	dim3 block;
	int thread;

	Block_Thread_Size(h_TriangleCountAfterClip, block, thread);
	Culling << <block, thread, thread * sizeof(int), S_CullingScanLineRange >> > (d_VertexPos, d_isDraw, d_DrawTriangleIdxPrefixSum);
	int blockCount = block.x * block.y;
	if (blockCount > 1)
	{
		CU_KoggeStoneScan_Block(d_DrawTriangleIdxPrefixSum, block, thread, h_TriangleCountAfterClip, blockCount, S_CullingScanLineRange);
	}
	hipMemcpy(&h_DrawTriangleCount, d_DrawTriangleIdxPrefixSum + h_TriangleCountAfterClip - 1, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(DrawTriangleCount), d_DrawTriangleIdxPrefixSum + h_TriangleCountAfterClip - 1, sizeof(int), 0, hipMemcpyDeviceToDevice, S_CullingScanLineRange);
	static int DrawTriangleArrayCount = 0;
	if (h_DrawTriangleCount > DrawTriangleArrayCount)
	{
		if (DrawTriangleArrayCount != 0)
		{
			hipFree(d_DrawVertexPos);
			hipFree(d_DrawVertexUV);
			hipFree(d_VerticalBottom);
			hipFree(d_VerticalHeightPrefixSum);
			hipFree(d_DrawTriangleIdxPrefixSumCom);
		}
		DrawTriangleArrayCount = h_DrawTriangleCount + (h_DrawTriangleCount >> 2);
		hipMalloc(&d_DrawVertexPos, sizeof(float4) * DrawTriangleArrayCount * 3);
		hipMalloc(&d_DrawVertexUV, sizeof(float2) * DrawTriangleArrayCount * 3);
		hipMalloc(&d_VerticalBottom, sizeof(int) * DrawTriangleArrayCount);
		hipMalloc(&d_VerticalHeightPrefixSum, sizeof(int) * DrawTriangleArrayCount);
		hipMalloc(&d_DrawTriangleIdxPrefixSumCom, sizeof(int) * DrawTriangleArrayCount);
	}
	SetDrawTriangleIdxPrefixCom << <block, thread, 0, S_CullingScanLineRange >> > (d_isDraw, d_DrawTriangleIdxPrefixSum, d_DrawTriangleIdxPrefixSumCom);
	Block_Thread_Size(h_DrawTriangleCount, block, thread);
	DrawVertexSetting_GetVerticalRange << <block, thread, thread * sizeof(int), S_CullingScanLineRange >> > (d_VertexPos, d_VertexUV, d_DrawVertexPos, d_DrawVertexUV, d_DrawTriangleIdxPrefixSumCom, d_VerticalBottom, d_VerticalHeightPrefixSum);
	
	blockCount = block.x * block.y;
	if (blockCount > 1)
	{
		CU_KoggeStoneScan_Block(d_VerticalHeightPrefixSum, block, thread, h_DrawTriangleCount, blockCount, S_CullingScanLineRange);
	}
	hipMemcpyToSymbolAsync(HIP_SYMBOL(DrawTriangleHeightCount), d_VerticalHeightPrefixSum + h_DrawTriangleCount - 1, sizeof(int), 0, hipMemcpyDeviceToDevice, S_CullingScanLineRange);
	hipMemcpy(&h_DrawTriangleHeightCount, d_VerticalHeightPrefixSum + h_DrawTriangleCount - 1, sizeof(int), hipMemcpyDeviceToHost);

	//return; //18~20

	static int DrawTriangleHeightArrayCount = 0;
	if (h_DrawTriangleHeightCount > DrawTriangleHeightArrayCount)
	{
		if (DrawTriangleHeightArrayCount != 0)
		{
			hipFree(d_DrawTriangleHeightTriangleIdx);
			hipFree(d_DrawTriangleHeight);
			hipFree(d_DrawTriangleHeightLeft);
			hipFree(d_DrawTriangleHeightRight);
			hipFree(d_DTHLeftUV);
			hipFree(d_DTHRightUV);
			hipFree(d_DTHLeftDepth);
			hipFree(d_DTHRightDepth);
			hipFree(d_FragmentCountPrefixSum);

		}
		DrawTriangleHeightArrayCount = h_DrawTriangleHeightCount + (h_DrawTriangleHeightCount >> 2);
		hipMalloc(&d_DrawTriangleHeightTriangleIdx, sizeof(int) * DrawTriangleHeightArrayCount);
		hipMalloc(&d_DrawTriangleHeight, sizeof(int) * DrawTriangleHeightArrayCount);
		hipMalloc(&d_DrawTriangleHeightLeft, sizeof(int) * DrawTriangleHeightArrayCount);
		hipMalloc(&d_DrawTriangleHeightRight, sizeof(int) * DrawTriangleHeightArrayCount);
		hipMalloc(&d_DTHLeftUV, sizeof(float2) * DrawTriangleHeightArrayCount);
		hipMalloc(&d_DTHRightUV, sizeof(float2) * DrawTriangleHeightArrayCount);
		hipMalloc(&d_DTHLeftDepth, sizeof(float) * DrawTriangleHeightArrayCount);
		hipMalloc(&d_DTHRightDepth, sizeof(float) * DrawTriangleHeightArrayCount);
		hipMalloc(&d_FragmentCountPrefixSum, sizeof(int) * DrawTriangleHeightArrayCount);
	}

	SetDrawTriangleHeightTriangleIdx << <block, thread, 0, S_CullingScanLineRange >> > (d_VerticalHeightPrefixSum, d_DrawTriangleHeightTriangleIdx);

	Block_Thread_Size(h_DrawTriangleHeightCount, block, thread);
	SetDrawTriangleHeight << <block, thread, 0, S_CullingScanLineRange >> > (d_VerticalBottom, d_VerticalHeightPrefixSum, d_DrawTriangleHeightTriangleIdx, d_DrawTriangleHeight);
	SetDrawTriangleHeightInfo << <block, thread, thread * sizeof(int), S_CullingScanLineRange >> > (d_DrawVertexPos, d_DrawVertexUV, d_DrawTriangleHeightTriangleIdx, d_DrawTriangleHeight,
		d_DrawTriangleHeightLeft, d_DrawTriangleHeightRight, d_DTHLeftUV, d_DTHRightUV, d_DTHLeftDepth, d_DTHRightDepth, d_FragmentCountPrefixSum, d_Log);

	blockCount = block.x * block.y;
	if (blockCount > 1)
	{
		CU_KoggeStoneScan_Block(d_FragmentCountPrefixSum, block, thread, h_DrawTriangleHeightCount, blockCount, S_CullingScanLineRange);
	}

	hipMemcpyToSymbolAsync(HIP_SYMBOL(FragmentCount), d_FragmentCountPrefixSum + h_DrawTriangleHeightCount - 1, sizeof(int), 0, hipMemcpyDeviceToDevice, S_CullingScanLineRange);
	hipMemcpy(&h_FragmentCount, d_FragmentCountPrefixSum + h_DrawTriangleHeightCount - 1, sizeof(int), hipMemcpyDeviceToHost);
	//return; //32~34
	static int FragmentArrayCount = 0;
	if (h_FragmentCount > FragmentArrayCount)
	{
		if (FragmentArrayCount != 0)
		{
			hipFree(d_FragmentDTHIdx);
		}
		FragmentArrayCount = h_FragmentCount + (h_FragmentCount >> 2);
		hipMalloc(&d_FragmentDTHIdx, sizeof(int) * FragmentArrayCount);
	}
	//return; //35
	SetFragmentDTHIdx << <block, thread, 0, S_CullingScanLineRange >> > (d_FragmentCountPrefixSum, d_FragmentDTHIdx);
	Block_Thread_Size(h_FragmentCount, block, thread);
	//return;
	SetBuffer << <block, thread, 0, S_CullingScanLineRange >> > (d_FragmentDTHIdx, d_FragmentCountPrefixSum, d_DrawTriangleHeight, d_DrawTriangleHeightLeft, d_DrawTriangleHeightRight, d_DTHLeftUV, d_DTHRightUV, d_DTHLeftDepth, d_DTHRightDepth,
		d_TexBuffer, d_TexWidth, d_TexHeight, d_ColorBuffer, d_DepthBuffer, d_Log);
	hipStreamSynchronize(S_CullingScanLineRange);
	hipStreamDestroy(S_CullingScanLineRange);
	hipMemcpy(h_Log, d_Log, sizeof(float) * 100, hipMemcpyDeviceToHost);
	float log0 = h_Log[0];
	float log1 = h_Log[1];
	float log2 = h_Log[2];
	float log3 = h_Log[3];
	float log4 = h_Log[4];
	float log5 = h_Log[5];
	int a = 0;
}


//�ﰢ���� �簢���� ���ϱ�
//��� �簢����
//

d_Matrix4x4* d_LocalToWorldMat;
Matrix4x4* h_LocalToWorldMat;
void CU_DrawCall(int materialIdx, vector<GameObject*> v_pGo_Visible)
{
	float startTime = clock();
	h_GoCount = v_pGo_Visible.size();
	static int GoArrayCount = 0;
	if (GoArrayCount < h_GoCount)
	{
		if (GoArrayCount != 0)
		{
			hipFree(d_MeshIdx);
			delete[] h_MeshIdx;
			hipFree(d_BCR);
			delete[] h_BCR;
			hipFree(d_LocalToWorldMat);
			delete[] h_LocalToWorldMat;
			hipFree(d_LocalToNDCMat);
		}
		GoArrayCount = h_GoCount + (h_GoCount >> 2);
		hipMalloc(&d_MeshIdx, sizeof(int) * GoArrayCount);
		h_MeshIdx = new int[GoArrayCount];
		hipMalloc(&d_BCR, sizeof(int) * GoArrayCount);
		h_BCR = new int[GoArrayCount];
		hipMalloc(&d_LocalToWorldMat, sizeof(d_Matrix4x4) * GoArrayCount);
		h_LocalToWorldMat = new Matrix4x4[GoArrayCount];
		hipMalloc(&d_LocalToNDCMat, sizeof(d_Matrix4x4) * GoArrayCount);
	}

	hipMemcpyToSymbolAsync(HIP_SYMBOL(MaterialIdx), &materialIdx, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbolAsync(HIP_SYMBOL(GoCount), &h_GoCount, sizeof(int), 0, hipMemcpyHostToDevice);
	for (int i = 0; i < h_GoCount; i++)
	{
		h_MeshIdx[i] = Engine::PInstance->GetMeshIdx(v_pGo_Visible[i]->GetMeshKey());
		h_LocalToWorldMat[i] = v_pGo_Visible[i]->GetTransform().GetModelingMatrix();
	}
	hipMemcpyAsync(d_MeshIdx, h_MeshIdx, sizeof(int) * h_GoCount, hipMemcpyHostToDevice);
	hipMemcpyAsync(d_LocalToWorldMat, h_LocalToWorldMat, sizeof(d_Matrix4x4) * h_GoCount, hipMemcpyHostToDevice);
	CU_ConstantMatMulMat(h_GoCount, h_WorldToNDCMat, d_LocalToWorldMat, d_LocalToNDCMat);
	Engine::PInstance->_TimeLog.CU_DrawCallSetting = clock() - startTime;
	startTime = clock();
	for (int i = 0; i < 1; i++)
	{
		CU_BoundCheckBox(); //h_BCR�� outside = 4, intersect = 2, inside = 1 ����
	}
	Engine::PInstance->_TimeLog.CU_BoundCheckBox = clock() - startTime;
	startTime = clock();
	for (int i = 0; i < 1; i++)
	{
		CU_VertexArraySetting();
	}
	Engine::PInstance->_TimeLog.CU_VertexArraySetting = clock() - startTime;
	startTime = clock();
	for (int i = 0; i < 1; i++)
	{
		CU_Culling_ScanLine_SetBuffer();
	}
	Engine::PInstance->_TimeLog.CU_Culling_ScanLine_SetBuffer = clock() - startTime;
	startTime = clock();
}

#pragma endregion


void CU_BufferCpy_Free(Color32* colorBuffer, float* depthBuffer)
{
	hipMemcpy(colorBuffer, d_ColorBuffer, sizeof(d_Color32) * h_ScreenSize, hipMemcpyDeviceToHost);
	hipMemcpy(depthBuffer, d_DepthBuffer, sizeof(float) * h_ScreenSize, hipMemcpyDeviceToHost);



}
